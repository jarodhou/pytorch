#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/nvrtc_stub/ATenNVRTC.h>

// Determine if the architecture supports rowwise scaled mm
// Currenlty failing on windows with: https://github.com/NVIDIA/cutlass/issues/1571
#if !defined(USE_ROCM) && !defined(_WIN32) && defined(CUDA_VERSION) && CUDA_VERSION >= 12000

#define BUILD_ROWWISE_FP8_KERNEL
#endif

#if defined(BUILD_ROWWISE_FP8_KERNEL)

// We are going to override the cuTensorMapEncodeTiled driver api with our lazy loader
static hipError_t CUDAAPI nvrtc_cuTensorMapEncodeTiled(
    CUtensorMap* tensorMap,
    CUtensorMapDataType tensorDataType,
    cuuint32_t tensorRank,
    void* globalAddress,
    const cuuint64_t* globalDim,
    const cuuint64_t* globalStrides,
    const cuuint32_t* boxDim,
    const cuuint32_t* elementStrides,
    CUtensorMapInterleave interleave,
    CUtensorMapSwizzle swizzle,
    CUtensorMapL2promotion l2Promotion,
    CUtensorMapFloatOOBfill oobFill) {
  return at::globalContext().getNVRTC().cuTensorMapEncodeTiled(
      tensorMap,
      tensorDataType,
      tensorRank,
      globalAddress,
      globalDim,
      globalStrides,
      boxDim,
      elementStrides,
      interleave,
      swizzle,
      l2Promotion,
      oobFill);
}


#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/half.h>
#include <cutlass/numeric_types.h>
#include <cutlass/trace.h>
#include <cutlass/util/host_tensor.h>

// Rename the global function symbol
#define cuTensorMapEncodeTiled nvrtc_cuTensorMapEncodeTiled
#include <cute/tensor.hpp>
#undef cuTensorMapEncodeTiled
// Set everything back to normal

#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/epilogue/collective/collective_builder.hpp>

#include <cute/atom/mma_atom.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>


namespace {

using DtypeScale = float;
using DtypeAccum = float;
using DtypeEpilogue = float;
using DtypeOutput = cutlass::bfloat16_t;

template <typename T>
struct identity {
  CUTLASS_HOST_DEVICE
  T operator()(T lhs) const {
    return lhs;
  }
};

using Multiply = cutlass::epilogue::fusion::Sm90Compute<
    cutlass::multiplies,
    DtypeEpilogue,
    DtypeEpilogue,
    cutlass::FloatRoundStyle::round_to_nearest>;

using Add = cutlass::epilogue::fusion::Sm90Compute<
    cutlass::plus,
    DtypeEpilogue,
    DtypeEpilogue,
    cutlass::FloatRoundStyle::round_to_nearest>;

using Cast = cutlass::epilogue::fusion::Sm90Compute<
    identity,
    DtypeOutput,
    DtypeEpilogue,
    cutlass::FloatRoundStyle::round_to_nearest>;

template <bool PingPong, bool FastAccum>
struct Schedule;

template <>
struct Schedule</*PingPong=*/false, /*FastAccum=*/false> {
  using type = cutlass::gemm::KernelTmaWarpSpecialized;
};

template <>
struct Schedule</*PingPong=*/true, /*FastAccum=*/false> {
  using type = cutlass::gemm::KernelTmaWarpSpecializedPingpong;
};

template <>
struct Schedule</*PingPong=*/false, /*FastAccum=*/true> {
  using type = cutlass::gemm::KernelTmaWarpSpecializedFP8FastAccum;
};

template <>
struct Schedule</*PingPong=*/true, /*FastAccum=*/true> {
  using type = cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
};

// Cutlass rowwise kernel
template <
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool PONG,
    bool FAST_ACCUM,
    typename DtypeA,
    typename DtypeB,
    typename DtypeBias>
void f8f8bf16_rowwise_impl(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    at::Tensor out) {
  int M = XQ.size(0);
  int N = WQ.size(1);
  int K = XQ.size(1);

  TORCH_CHECK(XQ.is_cuda() && XQ.is_contiguous());
  TORCH_CHECK(
      WQ.is_cuda() && WQ.ndimension() == 2 && WQ.stride(1) == WQ.size(0) &&
      WQ.stride(0) == 1);

  using LayoutInputA = cutlass::layout::RowMajor;
  constexpr int AlignmentInputA = 16 / sizeof(DtypeA);

  using LayoutInputB = cutlass::layout::ColumnMajor;
  constexpr int AlignmentInputB = 16 / sizeof(DtypeB);

  using LayoutOutput = cutlass::layout::RowMajor;
  constexpr int AlignmentOutput = 16 / sizeof(DtypeOutput);

  // Tag indicating the minimum SM that supports the intended feature
  using ArchTag = cutlass::arch::Sm90;
  using OperatorClass = cutlass::arch::OpClassTensorOp;
  // Threadblock-level tile size
  using TileShape =
      cute::Shape<cute::Int<TB_M>, cute::Int<TB_N>, cute::Int<TB_K>>;
  // Shape of the threadblocks in a cluster
  using ClusterShape =
      cute::Shape<cute::Int<TBS_M>, cute::Int<TBS_N>, cute::Int<TBS_K>>;

  // Implement rowwise scaling epilogue.
  constexpr int ColBcastStages = 0;
  constexpr int RowBcastStages = PONG ? 2 : 1;

  using XScale = cutlass::epilogue::fusion::
      Sm90ColBroadcast<ColBcastStages, TileShape, DtypeScale>;

  using WScale = cutlass::epilogue::fusion::
      Sm90RowBroadcast<RowBcastStages, TileShape, DtypeScale>;

  using Bias = cutlass::epilogue::fusion::
      Sm90RowBroadcast<RowBcastStages, TileShape, DtypeBias>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using EpilogueEVT = cutlass::epilogue::fusion::Sm90EVT<
      Cast,
      cutlass::epilogue::fusion::Sm90EVT<
          Add,
          Bias,
          cutlass::epilogue::fusion::Sm90EVT<
              Multiply,
              XScale,
              cutlass::epilogue::fusion::Sm90EVT<Multiply, WScale, Accum>>>>;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          TileShape,
          ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto,
          DtypeAccum,
          DtypeEpilogue,
          DtypeOutput,
          LayoutOutput,
          AlignmentOutput,
          DtypeOutput,
          LayoutOutput,
          AlignmentOutput,
          cutlass::epilogue::TmaWarpSpecialized,
          EpilogueEVT>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          DtypeA,
          LayoutInputA,
          AlignmentInputA,
          DtypeB,
          LayoutInputB,
          AlignmentInputB,
          DtypeAccum,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          typename Schedule<PONG, FAST_ACCUM>::type>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideInputA = typename Gemm::GemmKernel::StrideA;
  using StrideInputB = typename Gemm::GemmKernel::StrideB;
  using StrideOutput = typename Gemm::GemmKernel::StrideC;

  StrideInputA stride_a = cutlass::make_cute_packed_stride(
      StrideInputA{}, cute::make_shape(M, K, 1));
  StrideInputB stride_b = cutlass::make_cute_packed_stride(
      StrideInputB{}, cute::make_shape(N, K, 1));
  StrideOutput stride_output = cutlass::make_cute_packed_stride(
      StrideOutput{}, cute::make_shape(M, N, 1));

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K},
      {reinterpret_cast<DtypeA*>(XQ.data_ptr()),
       stride_a,
       reinterpret_cast<DtypeB*>(WQ.data_ptr()),
       stride_b},
      {{{{bias.has_value() ? reinterpret_cast<DtypeBias*>(bias->data_ptr())
                           : nullptr},
         {{reinterpret_cast<DtypeScale*>(x_scale.data_ptr())},
          {{reinterpret_cast<DtypeScale*>(w_scale.data_ptr())}}}}},
       reinterpret_cast<DtypeOutput*>(out.data_ptr()),
       stride_output,
       reinterpret_cast<DtypeOutput*>(out.data_ptr()),
       stride_output}};

  Gemm gemm;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  auto workspace = XQ.new_empty(
      {static_cast<int64_t>(workspace_size)},
      at::TensorOptions().dtype(at::kByte));

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(arguments, workspace.data_ptr());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm(at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

// FP8 Rowwise Cutlass kernel dispatch.
enum class KernelMode { Small, Large, Default };

KernelMode get_kernel_mode(at::Tensor XQ, at::Tensor WQ) {
  auto M = XQ.size(0);
  auto K = XQ.size(1);
  auto N = WQ.size(0);
  // Use a large kernel if at least two shapes are large....
  bool use_large_kernel =
      ((M >= 2048 && K >= 2048) || (M >= 2048 && N >= 2048) ||
       (K >= 2048 && N >= 2048));
  if (M <= 128 || N <= 128) {
    return KernelMode::Small;
  } else if (use_large_kernel) {
    return KernelMode::Large;
  } else {
    return KernelMode::Default;
  }
}

template <typename DtypeA, bool FastAccum, typename BiasDType>
void dispatch_fp8_rowwise_kernel(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    at::Tensor out) {
  KernelMode kernel = get_kernel_mode(XQ, WQ);
  if (kernel == KernelMode::Small) {
    return f8f8bf16_rowwise_impl<
        64,
        128,
        128,
        2,
        1,
        1,
        false,
        FastAccum,
        DtypeA,
        /*DtypeB=*/cutlass::float_e4m3_t,
        BiasDType>(XQ, WQ, x_scale, w_scale, bias, out);
  } else if (kernel == KernelMode::Large) {
    return f8f8bf16_rowwise_impl<
        128,
        128,
        128,
        2,
        1,
        1,
        true,
        FastAccum,
        DtypeA,
        /*DtypeB=*/cutlass::float_e4m3_t,
        BiasDType>(XQ, WQ, x_scale, w_scale, bias, out);
  } else {
    return f8f8bf16_rowwise_impl<
        128,
        128,
        128,
        1,
        2,
        1,
        false,
        FastAccum,
        DtypeA,
        /*DtypeB=*/cutlass::float_e4m3_t,
        BiasDType>(XQ, WQ, x_scale, w_scale, bias, out);
  }
}

} // namespace

#endif // !defined(USE_ROCM)

namespace at::cuda::detail {
void f8f8bf16_rowwise(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale, // FP32
    at::Tensor w_scale, // FP32
    std::optional<at::Tensor> bias, // BF16
    bool use_fast_accum,
    at::Tensor& out) {
#if defined(BUILD_ROWWISE_FP8_KERNEL)
  // Check datatypes.
  TORCH_CHECK(
      x_scale.dtype() == at::kFloat && w_scale.dtype() == at::kFloat,
      "Scale tensors must be float32.");
  if (bias.has_value()) {
    TORCH_CHECK(
        bias.value().dtype() == at::kFloat ||
            bias.value().dtype() == at::kBFloat16,
        "Bias type must be bfloat16 or float32 if provided.");
  }

  bool bf16_bias = bias.has_value() && bias->dtype() == at::kBFloat16;

  // Templatize based on input dtype.
  bool use_e5m2 = XQ.dtype() == at::kFloat8_e5m2;
  TORCH_CHECK(WQ.dtype() == at::kFloat8_e4m3fn, "For RowWise scaling the second input is required to be a float8_e4m3fn dtype.");

  if (bf16_bias) {
    if (use_fast_accum) {
      if (use_e5m2) {
        return dispatch_fp8_rowwise_kernel<
            cutlass::float_e5m2_t,
            true,
            cutlass::bfloat16_t>(XQ, WQ, x_scale, w_scale, bias, out);
      } else {
        return dispatch_fp8_rowwise_kernel<
            cutlass::float_e4m3_t,
            true,
            cutlass::bfloat16_t>(XQ, WQ, x_scale, w_scale, bias, out);
      }
    } else {
      if (use_e5m2) {
        return dispatch_fp8_rowwise_kernel<
            cutlass::float_e5m2_t,
            false,
            cutlass::bfloat16_t>(XQ, WQ, x_scale, w_scale, bias, out);
      } else {
        return dispatch_fp8_rowwise_kernel<
            cutlass::float_e4m3_t,
            false,
            cutlass::bfloat16_t>(XQ, WQ, x_scale, w_scale, bias, out);
      }
    }
  } else {
    if (use_fast_accum) {
      if (use_e5m2) {
        return dispatch_fp8_rowwise_kernel<cutlass::float_e5m2_t, true, float>(
            XQ, WQ, x_scale, w_scale, bias, out);
      } else {
        return dispatch_fp8_rowwise_kernel<cutlass::float_e4m3_t, true, float>(
            XQ, WQ, x_scale, w_scale, bias, out);
      }
    } else {
      if (use_e5m2) {
        return dispatch_fp8_rowwise_kernel<cutlass::float_e5m2_t, false, float>(
            XQ, WQ, x_scale, w_scale, bias, out);
      } else {
        return dispatch_fp8_rowwise_kernel<cutlass::float_e4m3_t, false, float>(
            XQ, WQ, x_scale, w_scale, bias, out);
      }
    }
  }
#else // BUILD_ROWWISE_FP8_KERNEL
  TORCH_CHECK(false, "Rowwise scaling is not currenlty supported on your device");
#endif
}

} // namespace at::cuda::detail
